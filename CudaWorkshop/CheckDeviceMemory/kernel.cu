﻿
#include "hip/hip_runtime.h"

#include <stdio.h>

// 디바이스 메모리 체크
void checkDeviceMemory(void)
{
	size_t free, total;
	hipMemGetInfo(&free, &total);
	printf("Device memory (free/total) = %lld/%lld bytes\n", free, total);
}

int main(void)
{
	int* dDataPointer;
	hipError_t errorCode;

	// 디바이스 메모리 할당
	checkDeviceMemory();
	errorCode = hipMalloc(&dDataPointer, sizeof(int) * 1024 * 1024);
	printf("hipMalloc - %s\n", hipGetErrorName(errorCode));

	// 디바이스 메모리 초기화
	checkDeviceMemory();
	errorCode = hipMemset(&dDataPointer, 0, sizeof(int) * 1024 * 1024);
	printf("hipMemset - %s\n", hipGetErrorName(errorCode));

	// 디바이스 메모리 해제
	errorCode = hipFree(dDataPointer);
	printf("hipFree - %s\n", hipGetErrorName(errorCode));
	checkDeviceMemory();
}