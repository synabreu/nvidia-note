﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void helloCUDA(void)
{
	printf("Hello, CUDA!\n");
}

int main(void)
{
	helloCUDA<<<1, 5>>>();
	return 0;
}